#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <fstream>
#include "helper.h"

using namespace std;

__global__ void countPopulation(int *result, float *lat, float *lon, int *pop,
    float lat_city, float lon_city, float kmRange) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    float phi1 = (90.f - lat[i]) * DEGREE_TO_RADIANS;
    float phi2 = (90.f - lat_city) * DEGREE_TO_RADIANS;

    float theta1 = lon[i] * DEGREE_TO_RADIANS;
    float theta2 = lon_city * DEGREE_TO_RADIANS;

    float cs = sin(phi1) * sin(phi2) * cos(theta1 - theta2) + cos(phi1) * cos(phi2);
    if (cs > 1) {
        cs = 1;
    } else if (cs < -1) {
        cs = -1;
    }

    float rez =  6371.f * acos(cs);
    if (rez <= kmRange) {
            atomicAdd(&result[blockIdx.x], pop[i]);
    }
}

// sampleFileIO demos reading test files and writing output
void my_sampleFileIO(float kmRange, const char* fileIn, const char* fileOut)
{
    string geon;
    float lat;
    float lon;
    int pop;
    int nr_of_cities = 0;

    float *host_lat_array = 0;
    float *host_lon_array = 0;
    int *host_pop_array = 0;

    host_lat_array = (float *) malloc(sizeof(float) * 1);
    host_lon_array = (float *) malloc(sizeof(float) * 1);
    host_pop_array = (int *) malloc(sizeof(int) * 1);

    float *device_lat_array = 0;
    float *device_lon_array = 0;
    int *device_pop_array = 0;
    int *device_result = 0;

    ifstream ifs(fileIn);
    ofstream ofs(fileOut);

    while(ifs >> geon >> lat >> lon >> pop)
    {
        host_lat_array[nr_of_cities] = lat;
        host_lon_array[nr_of_cities] = lon;
        host_pop_array[nr_of_cities] = pop;
        nr_of_cities++;

        host_lat_array  = (float *) realloc(host_lat_array, sizeof(float) * (nr_of_cities + 1));
        host_lon_array  = (float *) realloc(host_lon_array, sizeof(float) * (nr_of_cities + 1));
        host_pop_array  = (int *) realloc(host_pop_array, sizeof(int) * (nr_of_cities + 1));
    }

    hipMallocManaged(&device_lat_array, nr_of_cities * sizeof(float));
    if (device_lat_array == 0) {
        cout << "[HOST] Couldn't allocate memory\n";
        exit(-1);
    }

    hipMallocManaged(&device_lon_array, nr_of_cities * sizeof(float));
    if (device_lon_array == 0) {
        cout << "[HOST] Couldn't allocate memory\n";
        exit(-1);

    }

    hipMallocManaged(&device_pop_array, nr_of_cities * sizeof(int));
    if (device_pop_array == 0) {
        cout << "[HOST] Couldn't allocate memory\n";
        exit(-1);
    }

    hipMallocManaged(&device_result, nr_of_cities * sizeof(unsigned long long int));
    if (device_result == 0) {
        cout << "[HOST] Couldn't allocate memory\n";
        exit(-1);
    }
    
    hipMemset(device_result, 0, nr_of_cities);
    hipMemcpy(device_lat_array, host_lat_array, nr_of_cities * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_lon_array, host_lon_array, nr_of_cities * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_pop_array, host_pop_array, nr_of_cities * sizeof(int), hipMemcpyHostToDevice);

    const size_t block_size = 1024;
    size_t grid_size = nr_of_cities / block_size;
  
    if (nr_of_cities % block_size) 
        ++grid_size;

    for (int i = 0; i < nr_of_cities; i++) {
        countPopulation<<<grid_size, block_size>>>(device_result, device_lat_array,
            device_lon_array, device_pop_array, device_lat_array[i],
            device_lon_array[i], kmRange);
        hipDeviceSynchronize();

        ofs << device_result[0] << endl;
        hipMemset(device_result, 0, nr_of_cities);
    }

    ifs.close();
    ofs.close();

    free(host_lat_array);
    free(host_lon_array);
    free(host_pop_array);

    hipFree(device_lat_array);
    hipFree(device_lon_array);
    hipFree(device_pop_array);
    hipFree(device_result);
}


int main(int argc, char *argv[]) {
    DIE( argc == 1,
         "./accpop <kmrange1> <file1in> <file1out> ...");
    DIE( (argc - 1) % 3 != 0,
         "./accpop <kmrange1> <file1in> <file1out> ...");

    for(int argcID = 1; argcID < argc; argcID += 3) {
        float kmRange = atof(argv[argcID]);
        my_sampleFileIO(kmRange, argv[argcID + 1], argv[argcID + 2]);
    }
}